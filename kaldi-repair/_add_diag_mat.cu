
#include <hip/hip_runtime.h>
// cudamatrix: fixes to kernels that were using __syncthreads incorrectly (not sure if will fix a problem remi.fran6 was experiencing).

__global__ void _add_diag_mat_mat(
       double alpha, double* v, int v_dim, const double* M, int M_cols, int M_row_stride,
       int M_col_stride, const double *N, int N_row_stride, int N_col_stride,
       double beta) {

  // we actually assume blockDim.x == 256 here.
  // Each diagonal element of v is processed by "threads_per_element" threads.
  __shared__ double temp_data[256];
  int threads_per_element = 5;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int v_idx = i / threads_per_element,   // v_idx is the index into v that we are supposed to
      sub_idx = i % threads_per_element; // add to; 0 <= sub_idx < threads_per_element tells
                                         // us which block of elements we sum up.
  if (v_idx >= v_dim) return;

  double sum = 0.0;
  for (int j = sub_idx; j < M_cols; j += threads_per_element) {
    int M_index = v_idx * M_row_stride + j * M_col_stride,
        N_index = j * N_row_stride + v_idx * N_col_stride;
    sum += M[M_index] * N[N_index];
  }
  temp_data[threadIdx.x] = sum;

  // start_idx = threadIdx.x - sub_idx; // start of the position in temp_data
                                     // that we want to sum up.
  // The following is a tree-based reduction of the elements of temp_data from
  // start_idx to start_idx + threads_per_element - 1; our own index is "sub_idx".
  __syncthreads();
  int num_total_threads = threads_per_element;
  while (num_total_threads > 1) {
    int half_point = ((1 + num_total_threads) >> 1);
    if (sub_idx < half_point) {
      double temp = 0.0;
      if (sub_idx + half_point < num_total_threads) {
        temp = temp_data[threadIdx.x + half_point];
      }
      temp_data[threadIdx.x] += temp;
    }
    __syncthreads();
    num_total_threads = half_point;
  }
  if (sub_idx == 0) {
    v[v_idx] = beta * v[v_idx] + alpha * temp_data[threadIdx.x];
  }
}