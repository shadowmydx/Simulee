
#include <hip/hip_runtime.h>
#define N 15
#define B 2
#define T 32

__global__ void dl(int* in)
{
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < N)
  {
    if(tid % 2 == 0)
      in[tid]++;

    __syncthreads(); // ouch

    int sum = in[tid];
    if(tid > 0)
      sum += in[tid-1];
    if(tid < N - 1)
      sum += in[tid+1];
    in[tid] = sum / 3;
  }
}

// dl<<<B,T>>>(din);