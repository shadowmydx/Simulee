
#include <hip/hip_runtime.h>
#define N 64
#define B 1
#define T 64

__global__ void dl(int* in)
{
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // The warps in this block take different paths; the synctreads calls
  // will cause a deadlock.
  if(tid > 31)
  {
    if(tid % 2 == 0)
      in[tid]++;

    __syncthreads();

  }
  else {
    if(tid % 2 == 1)
      in[tid]--;
    
    __syncthreads();
  }
 }