
#include <hip/hip_runtime.h>
#define N 64
#define B 2
#define T 32

__global__ void dl(int* in)
{
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(blockIdx.x % 2 == 0)
  {
    if(tid % 2 == 0)
      in[tid]++;
    // Fine because conditional synchronization will
    // happen within a block.
    __syncthreads();

  }
  else {
    if(tid % 2 == 1)
      in[tid]--;
    
    __syncthreads();
  }
}