
#include <hip/hip_runtime.h>
__device__
double _sum_reduce(double buffer[]) {
   
  int nTotalThreads = blockDim.x;
  __syncthreads();
   
  while (nTotalThreads > 1) {
    int halfPoint = ((1 + nTotalThreads) >> 1);  
     
    if (threadIdx.x >= halfPoint) {  
       
      double temp = 0.0;
      if (threadIdx.x < nTotalThreads) {  
        temp = buffer[threadIdx.x];  
      }
      buffer[threadIdx.x - halfPoint] += temp;
    }
    __syncthreads();
    nTotalThreads = ((1 + nTotalThreads) >> 1);  
  }
   
  return buffer[0];
}