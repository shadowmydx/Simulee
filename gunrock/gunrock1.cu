
#include <hip/hip_runtime.h>
#define SizeT int
#define VertexId int

__global__ void Collect(
    const SizeT                 edges,
    const SizeT 		iter,
    const SizeT*    const 	flag,
    const VertexId* const	froms_data,
    const VertexId* const	tos_data,
    	  VertexId* 	        froms,
    	  VertexId* 	        tos,
    	  SizeT*     	        pos,
	  SizeT*		counts)
{
    SizeT x = blockIdx.x * blockDim.x + threadIdx.x;
    SizeT size = ((iter==0) ? pos[iter]:counts[0]) * (pos[iter+1]-pos[iter]);
    if(x>=0 && x<size*edges)
    {
	//SizeT a = x%(((iter==0)?pos[iter]:counts[0]) * edges);
	SizeT a = (x/edges%((iter==0)?pos[iter]:counts[0]))*edges+x%edges;
	SizeT b = pos[iter]+x/(edges*((iter==0)?pos[iter]:counts[0])); // edge iter+1 e_id

	if(flag[x/edges]>=1 && (x/edges==0 || flag[x/edges]>flag[x/edges-1]))
	{
// printf("large group:%d small group: %d  iter:%d froms_out[%d]:%d->tos_out[%d]:%d flag[%d]=%d\n",x/edges%(pos[iter+1]-pos[iter]), x/edges/(pos[iter+1]-pos[iter]),iter,a,froms[a],a,tos[a],x/edges,flag[x/edges]);
	    	VertexId from = froms[a];
	    	VertexId to = tos[a];
	    	//VertexId from = froms[x];
	    	//VertexId to = tos[x];
	    	__syncthreads();
		if(x%edges!=iter+1){
	    	froms[(flag[x/edges]-1)*edges+x%edges]=from;
		tos[(flag[x/edges]-1)*edges+x%edges]=to;}
		else{
		froms[(flag[x/edges]-1)*edges+iter+1] = froms_data[b];
		tos[(flag[x/edges]-1)*edges+iter+1] = tos_data[b];}
//printf("iter:%d 	froms[%d]:%d -> tos[%d]:%d	flag[%d]:%d\n",iter,(flag[x/edges]-1)*edges+x%edges,froms[(flag[x/edges]-1)*edges+x%edges],(flag[x/edges]-1)*edges+x%edges, tos[(flag[x/edges]-1)*edges+x%edges],x/edges,flag[x/edges]);
//printf("iter:%d 	froms[%d]:%d -> tos[%d]:%d	flag[%d]:%d\n",iter,(flag[x/edges]-1)*edges+x%edges,from,(flag[x/edges]-1)*edges+x%edges, to,x/edges,flag[x/edges]);
		counts[0] = flag[size-1];
	}
    } 
	
}