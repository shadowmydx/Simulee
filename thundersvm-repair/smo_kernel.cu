
#include <hip/hip_runtime.h>
// nu-svc fix bugs
__device__ int get_block_min(const float *values, int *index) {
    int tid = threadIdx.x;
    index[tid] = tid;
    __syncthreads();
    //block size is always the power of 2
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            if (values[index[tid + offset]] <= values[index[tid]]) {
                index[tid] = index[tid + offset];
            }
        }
        __syncthreads();
    }
    return index[0];
}

__host__ __device__ bool is_I_up(float a, float y, float C) {
    return (y > 0 && a < C) || (y < 0 && a > 0);
}

__host__ __device__ bool is_I_low(float a, float y, float C) {
    return (y > 0 && a > 0) || (y < 0 && a < C);
}

__global__ void
nu_smo_solve_kernel(const int *label, float *f_values, float *alpha, float *alpha_diff, const int *working_set,
                    int ws_size, float C, const float *k_mat_rows, const float *k_mat_diag, int row_len, float eps,
                    float *diff_and_bias) {
    //"row_len" equals to the number of instances in the original training dataset.
    //allocate shared memory
    __shared__ int shared_mem[256];
    int *f_idx2reduce = shared_mem; //temporary memory for reduction
    float *f_val2reduce = (float *) &f_idx2reduce[ws_size]; //f values used for reduction.
    float *alpha_i_diff = &f_val2reduce[ws_size]; //delta alpha_i
    float *alpha_j_diff = &alpha_i_diff[1];
    float *kd = &alpha_j_diff[1]; // diagonal elements for kernel matrix

    //index, f value and alpha for each instance
    int tid = threadIdx.x;
    int wsi = working_set[tid];
    kd[tid] = k_mat_diag[wsi];
    float y = label[wsi];
    float f = f_values[wsi];
    float a = alpha[wsi];
    float aold = a;
    __syncthreads();
    float local_eps = 0.0;
    int numOfIter = 0;
    while (1) {
        //select I_up (y=+1)
        if (y > 0 && a < C)
            f_val2reduce[tid] = f;
        else
            f_val2reduce[tid] = INFINITY;
       // __syncthreads();
        int ip = get_block_min(f_val2reduce, f_idx2reduce);
        float up_value_p = f_val2reduce[ip];
        float kIpwsI = k_mat_rows[row_len * ip + wsi];//K[i, wsi]
       // __syncthreads();


        float local_diff = up_value_p;

        if (numOfIter == 0) {
            local_eps = 0.1f * local_diff;
        }

        if (local_diff < local_eps) {
            alpha[wsi] = a;
            alpha_diff[tid] = -(a - aold) * y;
            if (tid == 0) {
                diff_and_bias[0] = local_diff;
            }
            break;
        }
    }
}

