
#include <hip/hip_runtime.h>
#define THREADS 256

__global__ void hamming_matcher_unroll(
    unsigned* out_idx,
    unsigned* out_dist,
    const unsigned max_dist)
{


    unsigned f = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tid = threadIdx.x;
	unsigned feat_len = 6, nquery = 6;

    __shared__ unsigned s_dist[THREADS];
    __shared__ unsigned s_idx[THREADS];

    s_dist[tid] = max_dist;
    s_idx[tid]  = 0xffffffff;
	unsigned ntrain = 64;
    bool valid_feat = (f < ntrain);
	

    for (unsigned j = 0; j < nquery; j++) {
        s_dist[tid] = max_dist;

        // Load one query feature that will be tested against all training
        // features in current block
        if (tid < feat_len && f < ntrain) {
            out_idx[tid] = tid * nquery + j;
        }
        __syncthreads();


        // Find best match in training features from block to the current
        // query feature
        if (tid < 32) {
            if (s_dist[tid + 128] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 128];
                s_idx[tid]  = s_idx[tid + 128];
            }
        }
        __syncthreads();
        if (tid < 16) {
            if (s_dist[tid + 64] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 64];
                s_idx[tid]  = s_idx[tid + 64];
            }
        }
        __syncthreads();
		if (tid < 8) {
            if (s_dist[tid + 32] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 32];
                s_idx[tid]  = s_idx[tid + 32];
            }
        }
        __syncthreads();

        // Store best match in training features from block to the current
        // query feature
        if (f < ntrain) {
            out_idx[j * gridDim.x + blockIdx.x] = s_dist[0];
            out_dist[j * gridDim.x + blockIdx.x]  = s_idx[0];
        }
    }
}


