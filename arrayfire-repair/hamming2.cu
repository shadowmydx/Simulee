
#include <hip/hip_runtime.h>
#define THREADS 256


__global__ void hamming_matcher(
    unsigned* out_idx,
    unsigned* out_dist,
    const unsigned max_dist,
    const unsigned feat_len)
{
    unsigned nquery = 6;
    unsigned ntrain = 6;

    unsigned f = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tid = threadIdx.x;

    __shared__ unsigned s_dist[THREADS];
    __shared__ unsigned s_idx[THREADS];


    s_dist[tid] = max_dist;
    s_idx[tid]  = 0xffffffff;

    bool valid_feat = (f < ntrain);

    for (unsigned j = 0; j < nquery; j++) {
        s_dist[tid] = max_dist;

        // Load one query feature that will be tested against all training
        // features in current block
        if (tid < feat_len && f < ntrain) {
            out_dist[tid] = tid * nquery + j;
        }
        __syncthreads();

        unsigned dist = 0;
        
        if (tid < 32) {
            if (s_dist[tid + 128] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 128];
                s_idx[tid]  = s_idx[tid + 128];
            }
        }
        __syncthreads();
        if (tid < 16) {
            if (s_dist[tid + 64] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 64];
                s_idx[tid]  = s_idx[tid + 64];
            }
        }
        __syncthreads();
        if (tid < 8) {
            if (s_dist[tid + 32] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + 32];
                s_idx[tid]  = s_idx[tid + 32];
            }
        }
        __syncthreads();

        // Store best match in training features from block to the current
        // query feature
        if (f < ntrain) {
            out_dist[j * gridDim.x + blockIdx.x] = s_dist[0];
            out_idx[j * gridDim.x + blockIdx.x]  = s_idx[0];
        }
    }
}