
#include <hip/hip_runtime.h>
    __device__ void warp_reduce(int *s_ptr, uint *s_idx)
    {
		int tidx = threadIdx.x;

        for (int n = 16; n >= 1; n >>= 1) {
            if (tidx < n) {
                int val1, val2;
				val1 = s_ptr[tidx];
				val2 = s_ptr[tidx + n];

                int idx1, idx2;
				idx1 = s_idx[tidx];
				idx2 = s_idx[tidx + n];

				s_ptr[tidx] = idx2;
				s_idx[tidx] = val2;
            }
        }
    }