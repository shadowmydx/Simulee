
#include <hip/hip_runtime.h>
#define THREADS 256

__global__ void select_matches(
    const unsigned* in_idx,
    const int* in_dist,
    const unsigned nfeat,
    const unsigned nelem,
    const int max_dist)
{
    unsigned f = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned sid = threadIdx.x * blockDim.y + threadIdx.y;

    __shared__ int s_dist[THREADS];
    __shared__ unsigned s_idx[THREADS];


    // Reduce best matches and find the best of them all
    for (unsigned i = blockDim.y / 2; i > 0; i >>= 1) {
        if (threadIdx.y < i) {
            int dist = s_dist[sid + i];
            if (dist < s_dist[sid]) {
                s_dist[sid] = dist;
                s_idx[sid]  = s_idx[sid + i];
            }
            __syncthreads();
        }
    }

}