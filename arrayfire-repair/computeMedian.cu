#include "hip/hip_runtime.h"
__global__ void computeMedian(
    const unsigned iterations)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned i = bid * blockDim.x + threadIdx.x;

    __shared__ float s_median[256];
    __shared__ unsigned s_idx[256];

    s_median[tid] = FLT_MAX;
    s_idx[tid] = 0;
    __syncthreads();

    if (i < iterations) {

        s_idx[tid] = i;
        s_median[tid] = m;
    }
}