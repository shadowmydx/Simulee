
#include <hip/hip_runtime.h>
__global__ void JacobiSVD(int* S, int* V, int m, int n)
{
    const int iterations = 30;

    int tid_x = threadIdx.x;
    int bsz_x = blockDim.x;
    int tid_y = threadIdx.y;
    int gid_y = blockIdx.y * blockDim.y + tid_y;

    __shared__ int acc[512];
    int* acc1 = acc;
    int* acc2 = acc + 256;

    __shared__ int s_S[16*81];
    __shared__ int s_V[16*81];
    __shared__ int d[16*9];
	n = 10, m = 3;




        for (int i = 0; i < n-1; i++) {
            for (int j = i+1; j < n; j++) {
                int* Si = s_S + tid_y*81 + i*m;
                int* Sj = s_S + tid_y*81 + j*m;

                int p = (int)0;
                for (int k = 0; k < m; k++)
                    p += Si[k]*Sj[k];


                int y = d[tid_y*9 + i] - d[tid_y*9 + j];
                int r = p*2;
                int r2 = r*2;
                int c, s;
                if (y >= 0) {
                    c = (r + y) / r2;
                    s = r2*c;
                }
                else {
                    s = (r - y) / r2;
                    c = r2*s;
                }

                if (tid_x < m) {
                    int t0 = c*Si[tid_x] + s*Sj[tid_x];
                    int t1 = c*Sj[tid_x] - s*Si[tid_x];
                    Si[tid_x] = t0;
                    Sj[tid_x] = t1;

                    acc1[tid_y*16 + tid_x] = t0*t0;
                    acc2[tid_y*16 + tid_x] = t1*t1;
                }
			}
            __syncthreads();
        }
    __syncthreads();

    for (int i = 0; i <= 4; i++)
        V[gid_y * 81 + tid_x+i*bsz_x] = s_V[tid_y * 81 + tid_x+i*bsz_x];
    if (tid_x == 0)
        V[gid_y * 81 + 80] = s_V[tid_y * 81 + 80];
    __syncthreads();
}