#include "hip/hip_runtime.h"
__device__ float block_reduce_sum(float val)
{
    __shared__ float data[256];

    unsigned idx = threadIdx.x * blockDim.x + threadIdx.y;

    data[idx] = val;
    __syncthreads();

    for (unsigned i = blockDim.y / 2; i > 0; i >>= 1)
    {
        if (threadIdx.y < i)
        {
            data[idx] += data[idx + i];
        }

        __syncthreads();
    }

    return data[threadIdx.x * blockDim.x];
}


__global__ void harris_response(
        float* score_out,
        float* size_out,
        const float* x_in,
        const float* y_in,
        const float* scl_in,
        const unsigned total_feat,
        float* image_ptr,
        const unsigned block_size,
        const float k_thr,
        const unsigned patch_size)
{
    unsigned f = blockDim.x * blockIdx.x + threadIdx.x;
	total_feat = 1000;
	block_size = 16;
    if (f < total_feat) {
        unsigned x, y;
        float scl = 1.f;
        if (scl > 0) {
            // Update x and y coordinates according to scale
            scl = scl_in[f];
            x = (x_in[f] * scl);
            y = (y_in[f] * scl);
        }
        else {
            x = (x_in[f]);
            y = (y_in[f]);
        }

        // Round feature size to nearest odd integer
        float size = 1.f;



        unsigned r = block_size / 2;

        float ixx = 0.f, iyy = 0.f, ixy = 0.f;
        unsigned block_size_sq = block_size * block_size;
        for (unsigned k = threadIdx.y; k < block_size_sq; k += blockDim.y) {
            int i = k / block_size - r;
            int j = k % block_size - r;

            // Calculate local x and y derivatives
            float ix = image_ptr[(x+i+1)] - image_ptr[(x+i-1)];
            float iy = image_ptr[(x+i)] - image_ptr[(x+i)];

            // Accumulate second order derivatives
            ixx += ix*ix;
            iyy += iy*iy;
            ixy += ix*iy;
        }
        __syncthreads();

        ixx = block_reduce_sum(ixx);
        iyy = block_reduce_sum(iyy);
        ixy = block_reduce_sum(ixy);
    }
}