
#include <hip/hip_runtime.h>
__global__ void computeDescriptor(
    float* desc_out,
    unsigned desc_len,
    unsigned histsz,
    const float* x_in,
    const float* y_in,
    const unsigned* layer_in,
    const float* response_in,
    const float* size_in,
    const float* ori_in,
    unsigned total_feat,
    const int d,
    const int n,
    const float scale,
    const float sigma,
    const int n_layers)
{
    const int tid_x = threadIdx.x;
    const int tid_y = threadIdx.y;
    const int bsz_x = blockDim.x;
    const int bsz_y = blockDim.y;

    const int f = blockIdx.y * bsz_y + tid_y;
	desc_len = 6;
	total_feat = 8;

    __shared__ float shrdMem[512];
    float* desc = shrdMem;
    float* accum = shrdMem + desc_len * histsz;
	histsz = 1;

    if (f < total_feat) {
		const int histlen = 16;
        const int hist_off = (tid_x % histsz) * desc_len;
		
		int i = tid_x;
		while (i < histlen*histsz) {
			desc[tid_y*histlen+i] = 0.f;
			i += bsz_x;
		}
		
        __syncthreads();
		int l = tid_x;
		while (l < desc_len*2) {
			desc[l    ] += desc[l+2*desc_len];
			l += bsz_x;
		}
		__syncthreads();
		l = tid_x;
		while (l < desc_len) {
			desc[l    ] += desc[l+desc_len];
			l += bsz_x;
		}

    }
}